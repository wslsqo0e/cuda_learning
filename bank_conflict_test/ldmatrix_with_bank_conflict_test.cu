#include "hip/hip_runtime.h"
#include "zee_utils.h"
#include "zee_ndarray.h"
#include <hip/hip_fp16.h>

union __align__(4) HalfPack {
    uint32_t u32;
    half h[2];
};

__global__ void ld_matrix_with_bank_conflict() {
    int tx = threadIdx.x + blockIdx.x*blockDim.x;
    int ty = threadIdx.y + blockIdx.y*blockDim.y;

    __shared__ half smem[32*64];  // 每行刚好是128Bytes，占据32个Bank

    if (tx==0 && ty==0) {
        for (int i = 0; i < 32*64; i++) {
            smem[i] = __float2half(static_cast<float>(i));
        }
    }
    __syncthreads();

    HalfPack regs[4];
    // 取smem左上角部分[16, 16]大小的数据
    half* smem_ptr = smem + (tx%16)*64 + (tx/16)*8;
    uint32_t smem_int_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
    // TO: R1(0, 1), R2(512, 513), R3(8, 9), R4(520, 521)
    // 分析其bank conflict 情况
    // 一次加载至多128Bytes，所以至少4个wavefronts, T0~T7, T8~T15, T16~T23, T24~T32
    // T0~T7，每次加载都有bank conflict (其加载的行地址都是偏差128Bytes的整数倍)
    // 所以T0~T7 需要8次 wavefronts
    // 其它thread group类似，总计需要 32次 wavefronts
    asm volatile("ldmatrix.sync.aligned.m8n8.x4.shared.b16 {%0, %1, %2, %3}, [%4];\n"
                 : "=r"(regs[0].u32), "=r"(regs[1].u32), "=r"(regs[2].u32), "=r"(regs[3].u32)
                 : "r"(smem_int_ptr));

    printf("[%d, %d]: R1(%.0f, %.0f), R2(%.0f, %.0f), R3(%.0f, %.0f), R4(%.0f, %.0f)\n", tx, ty,
          __half2float(regs[0].h[0]), __half2float(regs[0].h[1]),
          __half2float(regs[1].h[0]), __half2float(regs[1].h[1]),
          __half2float(regs[2].h[0]), __half2float(regs[2].h[1]),
          __half2float(regs[3].h[0]), __half2float(regs[3].h[1]));
}

int main() {
    printf("Hello World\n");
    ld_matrix_with_bank_conflict<<<1, 32>>>();
    hipDeviceSynchronize();
}
